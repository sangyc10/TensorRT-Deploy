#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "stdio.h"
#include <iostream>
#include "trt_preprocess.hpp"

namespace preprocess{

TransInfo    trans;
AffineMatrix affine_matrix;

void warpaffine_init(int srcH, int srcW, int tarH, int tarW){
    trans.src_h = srcH;
    trans.src_w = srcW;
    trans.tar_h = tarH;
    trans.tar_w = tarW;
    affine_matrix.init(trans);
}

__host__ __device__ void affine_transformation(
    float trans_matrix[6], 
    int src_x, int src_y, 
    float* tar_x, float* tar_y)
{
    *tar_x = trans_matrix[0] * src_x + trans_matrix[1] * src_y + trans_matrix[2];
    *tar_y = trans_matrix[3] * src_x + trans_matrix[4] * src_y + trans_matrix[5];
}

__global__ void nearest_BGR2RGB_nhwc2nchw_norm_kernel(
    float* tar, uint8_t* src, 
    int tarW, int tarH, 
    int srcW, int srcH,
    float scaled_w, float scaled_h,
    float* d_mean, float* d_std) 
{
    //->resized之后的图tar上的坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //->计算最近坐标
    int src_y = floor((float)y * scaled_h);
    int src_x = floor((float)x * scaled_w);

    if (src_x < 0 || src_y < 0 || src_x > srcW || src_y > srcH) {
        //->越界的部分，不进行计算
    } else {
        //->计算tar中对应坐标的索引
        int tarIdx  = y * tarW + x;
        int tarArea = tarW * tarH;

        //->计算src中最近邻坐标的索引
        int srcIdx = (src_y * srcW + src_x) * 3;

        //->实现nearest beighbour的resize + BGR2RGB + nhwc2nchw + norm
        tar[tarIdx + tarArea * 0] = (src[srcIdx + 2] / 255.0f - d_mean[2]) / d_std[2];
        tar[tarIdx + tarArea * 1] = (src[srcIdx + 1] / 255.0f - d_mean[1]) / d_std[1];
        tar[tarIdx + tarArea * 2] = (src[srcIdx + 0] / 255.0f - d_mean[0]) / d_std[0];
    }
}

__global__ void bilinear_BGR2RGB_nhwc2nchw_norm_kernel(
    float* tar, uint8_t* src, 
    int tarW, int tarH, 
    int srcW, int srcH, 
    float scaled_w, float scaled_h,
    float* d_mean, float* d_std) 
{

    //->resized之后的图tar上的坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //->计算x,y映射到原图时最近的4个坐标
    int src_y1 = floor((y + 0.5) * scaled_h - 0.5);
    int src_x1 = floor((x + 0.5) * scaled_w - 0.5);
    int src_y2 = src_y1 + 1;
    int src_x2 = src_x1 + 1;

    if (src_y1 < 0 || src_x1 < 0 || src_y2 > srcH || src_x2 > srcW) {
        //->对于越界的坐标不进行计算
    } else {
        //->计算原图上的坐标(浮点类型)在0~1之间的值
        float th   = ((y + 0.5) * scaled_h - 0.5) - src_y1;
        float tw   = ((x + 0.5) * scaled_w - 0.5) - src_x1;

        //->计算面积
        float a1_1 = (1.0 - tw) * (1.0 - th);  
        float a1_2 = tw * (1.0 - th);         
        float a2_1 = (1.0 - tw) * th;         
        float a2_2 = tw * th;                 

        //->计算4个坐标所对应的索引
        int srcIdx1_1 = (src_y1 * srcW + src_x1) * 3; 
        int srcIdx1_2 = (src_y1 * srcW + src_x2) * 3;  
        int srcIdx2_1 = (src_y2 * srcW + src_x1) * 3;  
        int srcIdx2_2 = (src_y2 * srcW + src_x2) * 3;  

        //->计算resized之后的图的索引
        int tarIdx    = y * tarW  + x;
        int tarArea   = tarW * tarH;

        //->实现bilinear interpolation的resize + BGR2RGB + NHWC2NCHW normalization
        tar[tarIdx + tarArea * 0] = 
            (round((a1_1 * src[srcIdx1_1 + 2] + 
                   a1_2 * src[srcIdx1_2 + 2] +
                   a2_1 * src[srcIdx2_1 + 2] +
                   a2_2 * src[srcIdx2_2 + 2])) / 255.0f - d_mean[2]) / d_std[2];

        tar[tarIdx + tarArea * 1] = 
            (round((a1_1 * src[srcIdx1_1 + 1] + 
                   a1_2 * src[srcIdx1_2 + 1] +
                   a2_1 * src[srcIdx2_1 + 1] +
                   a2_2 * src[srcIdx2_2 + 1])) / 255.0f - d_mean[1]) / d_std[1];

        tar[tarIdx + tarArea * 2] = 
            (round((a1_1 * src[srcIdx1_1 + 0] + 
                   a1_2 * src[srcIdx1_2 + 0] +
                   a2_1 * src[srcIdx2_1 + 0] +
                   a2_2 * src[srcIdx2_2 + 0])) / 255.0f - d_mean[0]) / d_std[0];

    }
}

__global__ void bilinear_BGR2RGB_nhwc2nchw_shift_norm_kernel(
    float* tar, uint8_t* src, 
    int tarW, int tarH, 
    int srcW, int srcH, 
    float scaled_w, float scaled_h,
    float* d_mean, float* d_std) 
{
    //->resized后的图tar上的坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //->计算x,y映射到原图时最近的4个坐标
    int src_y1 = floor((y + 0.5) * scaled_h - 0.5);
    int src_x1 = floor((x + 0.5) * scaled_w - 0.5);
    int src_y2 = src_y1 + 1;
    int src_x2 = src_x1 + 1;

    if (src_y1 < 0 || src_x1 < 0 || src_y2 > srcH || src_x2 > srcW) {
        //->对于越界的坐标不进行计算
    } else {
        //->计算原图上的坐标(浮点类型)在0~1之间的值
        float th   = (float)y * scaled_h - src_y1;
        float tw   = (float)x * scaled_w - src_x1;

        //->计算面积
        float a1_1 = (1.0 - tw) * (1.0 - th);
        float a1_2 = tw * (1.0 - th);          
        float a2_1 = (1.0 - tw) * th;         
        float a2_2 = tw * th;                 

        //->计算4个坐标所对应的索引
        int srcIdx1_1 = (src_y1 * srcW + src_x1) * 3;  
        int srcIdx1_2 = (src_y1 * srcW + src_x2) * 3;  
        int srcIdx2_1 = (src_y2 * srcW + src_x1) * 3;  
        int srcIdx2_2 = (src_y2 * srcW + src_x2) * 3;  

        //->计算原图在目标图中的x, y方向上的偏移量
        y = y - int(srcH / (scaled_h * 2)) + int(tarH / 2);
        x = x - int(srcW / (scaled_w * 2)) + int(tarW / 2);

        //->计算resized之后的图的索引
        int tarIdx    = (y * tarW  + x) * 3;
        int tarArea   = tarW * tarH;

        //->实现bilinear interpolation + BGR2RGB + shift + nhwc2nchw
        tar[tarIdx + tarArea * 0] = 
            (round((a1_1 * src[srcIdx1_1 + 2] + 
                   a1_2 * src[srcIdx1_2 + 2] +
                   a2_1 * src[srcIdx2_1 + 2] +
                   a2_2 * src[srcIdx2_2 + 2])) / 255.0f - d_mean[2]) / d_std[2];

        tar[tarIdx + tarArea * 1] = 
            (round((a1_1 * src[srcIdx1_1 + 1] + 
                   a1_2 * src[srcIdx1_2 + 1] +
                   a2_1 * src[srcIdx2_1 + 1] +
                   a2_2 * src[srcIdx2_2 + 1])) / 255.0f - d_mean[1]) / d_std[1];

        tar[tarIdx + tarArea * 2] = 
            (round((a1_1 * src[srcIdx1_1 + 0] + 
                   a1_2 * src[srcIdx1_2 + 0] +
                   a2_1 * src[srcIdx2_1 + 0] +
                   a2_2 * src[srcIdx2_2 + 0])) / 255.0f - d_mean[0]) / d_std[0];
    }
}


__global__ void nearest_BGR2RGB_nhwc2nchw_kernel(
    float* tar, uint8_t* src, 
    int tarW, int tarH, 
    int srcW, int srcH,
    float scaled_w, float scaled_h)
{
    //->resized之后的图tar上的坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //->计算最近坐标
    int src_y = floor((float)y * scaled_h);
    int src_x = floor((float)x * scaled_w);

    if (src_x < 0 || src_y < 0 || src_x > srcW || src_y > srcH) {
        //->越界的部分，不进行计算
    } else {
        //->计算tar中对应坐标的索引
        int tarIdx  = y * tarW + x;
        int tarArea = tarW * tarH;

        //->计算src中最近邻坐标的索引
        int srcIdx = (src_y * srcW + src_x) * 3;

        //->实现nearest beighbour的resize + BGR2RGB + nhwc2nchw + norm
        tar[tarIdx + tarArea * 0] = src[srcIdx + 2] / 255.0f;
        tar[tarIdx + tarArea * 1] = src[srcIdx + 1] / 255.0f;
        tar[tarIdx + tarArea * 2] = src[srcIdx + 0] / 255.0f;
    }
}

__global__ void bilinear_BGR2RGB_nhwc2nchw_kernel(
    float* tar, uint8_t* src, 
    int tarW, int tarH, 
    int srcW, int srcH, 
    float scaled_w, float scaled_h)
{

    //->resized之后的图tar上的坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //->计算x,y映射到原图时最近的4个坐标
    int src_y1 = floor((y + 0.5) * scaled_h - 0.5);
    int src_x1 = floor((x + 0.5) * scaled_w - 0.5);
    int src_y2 = src_y1 + 1;
    int src_x2 = src_x1 + 1;

    if (src_y1 < 0 || src_x1 < 0 || src_y2 > srcH || src_x2 > srcW) {
        //->对于越界的坐标不进行计算
    } else {
        // ->计算原图上的坐标(浮点类型)在0~1之间的值
        float th   = ((y + 0.5) * scaled_h - 0.5) - src_y1;
        float tw   = ((x + 0.5) * scaled_w - 0.5) - src_x1;

        //->计算面积
        float a1_1 = (1.0 - tw) * (1.0 - th);  
        float a1_2 = tw * (1.0 - th);         
        float a2_1 = (1.0 - tw) * th;         
        float a2_2 = tw * th;                 

        //->计算4个坐标所对应的索引
        int srcIdx1_1 = (src_y1 * srcW + src_x1) * 3;  
        int srcIdx1_2 = (src_y1 * srcW + src_x2) * 3;  
        int srcIdx2_1 = (src_y2 * srcW + src_x1) * 3;  
        int srcIdx2_2 = (src_y2 * srcW + src_x2) * 3;  

        //->计算resized之后的图的索引
        int tarIdx    = y * tarW  + x;
        int tarArea   = tarW * tarH;

        //->实现bilinear interpolation的resize + BGR2RGB + NHWC2NCHW normalization
        tar[tarIdx + tarArea * 0] = 
            round((a1_1 * src[srcIdx1_1 + 2] + 
                   a1_2 * src[srcIdx1_2 + 2] +
                   a2_1 * src[srcIdx2_1 + 2] +
                   a2_2 * src[srcIdx2_2 + 2])) / 255.0f;

        tar[tarIdx + tarArea * 1] = 
            round((a1_1 * src[srcIdx1_1 + 1] + 
                   a1_2 * src[srcIdx1_2 + 1] +
                   a2_1 * src[srcIdx2_1 + 1] +
                   a2_2 * src[srcIdx2_2 + 1])) / 255.0f;

        tar[tarIdx + tarArea * 2] = 
            round((a1_1 * src[srcIdx1_1 + 0] + 
                   a1_2 * src[srcIdx1_2 + 0] +
                   a2_1 * src[srcIdx2_1 + 0] +
                   a2_2 * src[srcIdx2_2 + 0])) / 255.0f;

    }
}

__global__ void bilinear_BGR2RGB_nhwc2nchw_shift_kernel(
    float* tar, uint8_t* src, 
    int tarW, int tarH, 
    int srcW, int srcH, 
    float scaled_w, float scaled_h)
{
    //->resized之后的图tar上的坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;


    //->计算x,y映射到原图时最近的4个坐标
    int src_y1 = floor((y + 0.5) * scaled_h - 0.5);
    int src_x1 = floor((x + 0.5) * scaled_w - 0.5);
    int src_y2 = src_y1 + 1;
    int src_x2 = src_x1 + 1;

    if (src_y1 < 0 || src_x1 < 0 || src_y2 > srcH || src_x2 > srcW) {
        //->越界的坐标，不进行计算
    } else {
        //->计算原图上的坐标(浮点类型)在0~1之间的值
        float th   = (float)y * scaled_h - src_y1;
        float tw   = (float)x * scaled_w - src_x1;

        //->计算面积
        float a1_1 = (1.0 - tw) * (1.0 - th);  
        float a1_2 = tw * (1.0 - th);         
        float a2_1 = (1.0 - tw) * th;          
        float a2_2 = tw * th;                 

        //->计算4个坐标所对应的索引
        int srcIdx1_1 = (src_y1 * srcW + src_x1) * 3;  
        int srcIdx1_2 = (src_y1 * srcW + src_x2) * 3;  
        int srcIdx2_1 = (src_y2 * srcW + src_x1) * 3;  
        int srcIdx2_2 = (src_y2 * srcW + src_x2) * 3;  

        //->计算原图在目标图中的x, y方向上的偏移量
        y = y - int(srcH / (scaled_h * 2)) + int(tarH / 2);
        x = x - int(srcW / (scaled_w * 2)) + int(tarW / 2);

        //->计算resized之后的图的索引
        int tarIdx    = y * tarW  + x;
        int tarArea   = tarW * tarH;

        //->实现bilinear interpolation + BGR2RGB + shift + nhwc2nchw
        tar[tarIdx + tarArea * 0] = 
            round((a1_1 * src[srcIdx1_1 + 2] + 
                   a1_2 * src[srcIdx1_2 + 2] +
                   a2_1 * src[srcIdx2_1 + 2] +
                   a2_2 * src[srcIdx2_2 + 2])) / 255.0f;

        tar[tarIdx + tarArea * 1] = 
            round((a1_1 * src[srcIdx1_1 + 1] + 
                   a1_2 * src[srcIdx1_2 + 1] +
                   a2_1 * src[srcIdx2_1 + 1] +
                   a2_2 * src[srcIdx2_2 + 1])) / 255.0f;

        tar[tarIdx + tarArea * 2] = 
            round((a1_1 * src[srcIdx1_1 + 0] + 
                   a1_2 * src[srcIdx1_2 + 0] +
                   a2_1 * src[srcIdx2_1 + 0] +
                   a2_2 * src[srcIdx2_2 + 0])) / 255.0f;
    }
}

__global__ void warpaffine_BGR2RGB_kernel(
    float* tar, uint8_t* src, 
    TransInfo trans,
    AffineMatrix affine_matrix)
{
    float src_x, src_y;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 640 || y >= 640) return;                  //->防止越界
    
    affine_transformation(affine_matrix.reverse, x + 0.5, y + 0.5, &src_x, &src_y);

    int src_x1 = floor(src_x - 0.5);
    int src_y1 = floor(src_y - 0.5);
    int src_x2 = src_x1 + 1;
    int src_y2 = src_y1 + 1;

    if (src_y1 < 0 || src_x1 < 0 || src_y1 > trans.src_h || src_x1 > trans.src_w) {
    } else {
        float tw   = src_x - src_x1;
        float th   = src_y - src_y1;

        float a1_1 = (1.0 - tw) * (1.0 - th);
        float a1_2 = tw * (1.0 - th);
        float a2_1 = (1.0 - tw) * th;
        float a2_2 = tw * th;

        int srcIdx1_1 = (src_y1 * trans.src_w + src_x1) * 3;
        int srcIdx1_2 = (src_y1 * trans.src_w + src_x2) * 3;
        int srcIdx2_1 = (src_y2 * trans.src_w + src_x1) * 3;
        int srcIdx2_2 = (src_y2 * trans.src_w + src_x2) * 3;

        int tarIdx    = y * trans.tar_w  + x;
        int tarArea   = trans.tar_w * trans.tar_h;

        tar[tarIdx + tarArea * 0] = 
            round((a1_1 * src[srcIdx1_1 + 2] + 
                   a1_2 * src[srcIdx1_2 + 2] +
                   a2_1 * src[srcIdx2_1 + 2] +
                   a2_2 * src[srcIdx2_2 + 2])) / 255.0f;

        tar[tarIdx + tarArea * 1] = 
            round((a1_1 * src[srcIdx1_1 + 1] + 
                   a1_2 * src[srcIdx1_2 + 1] +
                   a2_1 * src[srcIdx2_1 + 1] +
                   a2_2 * src[srcIdx2_2 + 1])) / 255.0f;

        tar[tarIdx + tarArea * 2] = 
            round((a1_1 * src[srcIdx1_1 + 0] + 
                   a1_2 * src[srcIdx1_2 + 0] +
                   a2_1 * src[srcIdx2_1 + 0] +
                   a2_2 * src[srcIdx2_2 + 0])) / 255.0f;
    }
}

void resize_bilinear_gpu(
    float* d_tar, uint8_t* d_src, 
    int tarW, int tarH, 
    int srcW, int srcH, 
    float* d_mean, float* d_std,
    tactics tac) 
{
    dim3 dimBlock(32, 32, 1);
    dim3 dimGrid(tarW / 32 + 1, tarH / 32 + 1, 1);
   
    float scaled_h = (float)srcH / tarH;
    float scaled_w = (float)srcW / tarW;
    float scale = (scaled_h > scaled_w ? scaled_h : scaled_w);

    switch (tac) {
    case tactics::GPU_NEAREST:
        nearest_BGR2RGB_nhwc2nchw_norm_kernel 
                <<<dimGrid, dimBlock>>>
                (d_tar, d_src, tarW, tarH, srcW, srcH, scaled_w, scaled_h, d_mean, d_std);
        break;
    case tactics::GPU_NEAREST_CENTER:
        nearest_BGR2RGB_nhwc2nchw_norm_kernel 
                <<<dimGrid, dimBlock>>>
                (d_tar, d_src, tarW, tarH, srcW, srcH, scale, scale, d_mean, d_std);
        break;
    case tactics::GPU_BILINEAR:
        bilinear_BGR2RGB_nhwc2nchw_norm_kernel 
                <<<dimGrid, dimBlock>>> 
                (d_tar, d_src, tarW, tarH, srcW, srcH, scaled_w, scaled_h, d_mean, d_std);
        break;
    case tactics::GPU_BILINEAR_CENTER:
        bilinear_BGR2RGB_nhwc2nchw_shift_norm_kernel 
                <<<dimGrid, dimBlock>>> 
                (d_tar, d_src, tarW, tarH, srcW, srcH, scale, scale, d_mean, d_std);
        break;
    default:
        LOGE("ERROR: Wrong GPU resize tactics selected. Program terminated");
        exit(1);
    }
}

void resize_bilinear_gpu(
    float* d_tar, uint8_t* d_src, 
    int tarW, int tarH, 
    int srcW, int srcH, 
    tactics tac) 
{
    dim3 dimBlock(32, 32, 1);
    dim3 dimGrid(tarW / 32 + 1, tarH / 32 + 1, 1);
   
    //scaled resize
    float scaled_h = (float)srcH / tarH;
    float scaled_w = (float)srcW / tarW;
    float scale = (scaled_h > scaled_w ? scaled_h : scaled_w);

    switch (tac) {
    case tactics::GPU_NEAREST:
        nearest_BGR2RGB_nhwc2nchw_kernel <<<dimGrid, dimBlock>>>
                (d_tar, d_src, tarW, tarH, srcW, srcH, scaled_w, scaled_h);
        break;
    case tactics::GPU_NEAREST_CENTER:
        nearest_BGR2RGB_nhwc2nchw_kernel <<<dimGrid, dimBlock>>>
                (d_tar, d_src, tarW, tarH, srcW, srcH, scale, scale);
        break;
    case tactics::GPU_BILINEAR:
        bilinear_BGR2RGB_nhwc2nchw_kernel <<<dimGrid, dimBlock>>> 
                (d_tar, d_src, tarW, tarH, srcW, srcH, scaled_w, scaled_h);
        break;
    case tactics::GPU_BILINEAR_CENTER:
        bilinear_BGR2RGB_nhwc2nchw_shift_kernel <<<dimGrid, dimBlock>>> 
                (d_tar, d_src, tarW, tarH, srcW, srcH, scale, scale);
        break;
    case tactics::GPU_WARP_AFFINE:
        warpaffine_init(srcH, srcW, tarH, tarW);
        warpaffine_BGR2RGB_kernel <<<dimGrid, dimBlock>>> 
                (d_tar, d_src, trans, affine_matrix);
        break;
    default:
        LOGE("ERROR: Wrong GPU resize tactics selected. Program terminated");
        exit(1);
    }
}

} //->namespace preprocess
